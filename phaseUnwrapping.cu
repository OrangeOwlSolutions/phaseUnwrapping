#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // for C++
#include <cmath>

#include <thrust\scan.h>
#include <thrust\execution_policy.h>
#include <thrust\transform.h>
#include <thrust\device_ptr.h>

#include "Utilities.cuh"

#define BLOCKSIZE	256

/**************************************/
/* 1D PHASE UNWRAPPING - HOST VERSION */
/**************************************/
template<class T>
void unwrap1D_host(T * __restrict p, const int N) {

	T dp;
	T dps;
	T *dp_corr	= (T *)malloc(N * sizeof(T));
	T *cumsum	= (T *)malloc(N * sizeof(T));
	T cutoff	= M_PI;               /* default value in matlab */

	for (int j = 0; j < N - 1; j++) {

		// --- Incremental phase variation --- dp = diff(p, 1, 1);
		dp = p[j + 1] - p[j];

		// --- Equivalent phase variation in [-pi, pi] --- dps = mod(dp + dp, 2 * pi) - pi;
		dps = (dp + M_PI) - floor((dp + M_PI) / (2 * M_PI)) * (2 * M_PI) - M_PI;

		// --- Preserve variation sign for +pi vs. -pi --- dps(dps == pi & dp > 0, :) = pi;
		if ((dps == -M_PI) && (dp > 0)) dps = M_PI;

		// --- Incremental phase correction --- dp_corr = dps - dp;
		dp_corr[j] = dps - dp;

		// --- Ignore correction when incremental variation is smaller than cutoff --- dp_corr(abs(dp) < cutoff, :) = 0;
		if (fabs(dp) < cutoff) dp_corr[j] = (T)0;

	}

	// --- Find cumulative sum of deltas --- cumsum = cumsum(dp_corr, 1);
	thrust::inclusive_scan(thrust::host, dp_corr, dp_corr + N - 1, cumsum);

	// --- Integrate corrections and add to P to produce smoothed phase values --- p(2 : m, :) = p(2 : m, :) + cumsum(dp_corr, 1);
	thrust::transform(thrust::host, p + 1, p + N, cumsum, p + 1, thrust::plus<T>());

}

template void unwrap1D_host<float> (float  * __restrict, const int);
template void unwrap1D_host<double>(double * __restrict, const int);

/*****************************************/
/* 1D PHASE UNWRAPPING - GLOBAL FUNCTION */
/*****************************************/
template<class T>
__global__ void unwrap_1D_global(T * __restrict__ p, T * __restrict__ dp_corr, T * __restrict__ cumsum, T cutoff, const int N) {

	const int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= N - 1) return;

	T dp, dps;

	// --- Incremental phase variation --- dp = diff(p, 1, 1);
	dp = p[tid + 1] - p[tid];

	// --- Equivalent phase variation in [-pi, pi] --- dps = mod(dp + dp, 2 * pi) - pi;
	dps = (dp + M_PI) - floor((dp + M_PI) / (2 * M_PI)) * (2 * M_PI) - M_PI;

	// --- Preserve variation sign for +pi vs. -pi --- dps(dps == pi & dp > 0, :) = pi;
	if ((dps == -M_PI) && (dp > 0)) dps = M_PI;

	// --- Incremental phase correction --- dp_corr = dps - dp;
	dp_corr[tid] = dps - dp;

	// --- Ignore correction when incremental variation is smaller than cutoff --- dp_corr(abs(dp) < cutoff, :) = 0;
	if (fabs(dp) < cutoff) dp_corr[tid] = (T)0;

}

/****************************************/
/* 1D PHASE UNWRAPPING - DEVICE VERSION */
/****************************************/
template<class T>
void unwrap1D_device(T * __restrict__ p, const int N) {

	T *dp_corr;	gpuErrchk(hipMalloc((void**)&dp_corr, N * sizeof(T)));
	T *cumsum;	gpuErrchk(hipMalloc((void**)&cumsum, N * sizeof(T)));
	T cutoff = M_PI;               /* default value in matlab */

	unwrap_1D_global << <iDivUp(N, BLOCKSIZE), BLOCKSIZE >> >(p, dp_corr, cumsum, cutoff, N);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	// --- Find cumulative sum of deltas --- cumsum = cumsum(dp_corr, 1);
	thrust::inclusive_scan(thrust::device_pointer_cast(dp_corr), thrust::device_pointer_cast(dp_corr) + N - 1, thrust::device_pointer_cast(cumsum));

	// --- Integrate corrections and add to P to produce smoothed phase values --- p(2 : m, :) = p(2 : m, :) + cumsum(dp_corr, 1);
	thrust::transform(thrust::device_pointer_cast(p) + 1, thrust::device_pointer_cast(p) + N, thrust::device_pointer_cast(cumsum), thrust::device_pointer_cast(p) + 1, thrust::plus<T>());

	gpuErrchk(hipFree(dp_corr));
	gpuErrchk(hipFree(cumsum));

}

template void unwrap1D_device<float> (float  * __restrict, const int);
template void unwrap1D_device<double>(double * __restrict, const int);
